#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float Ashared[TILE_SIZE][TILE_SIZE];
    __shared__ float Bshared[TILE_SIZE][TILE_SIZE];

    // Calculate the X and Y of blocks and threads
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int threadX = threadIdx.x;
    int threadY = threadIdx.y;

    // Compute the row and column of the targeted product element
    int threadRow = blockY * TILE_SIZE + threadY;
    int threadColumn = blockX * TILE_SIZE + threadX;

    // Store the value
    float calcVal = 0;

    // I think k is the correct value to use here, because it's the shared dimension
    for (int a = 0; a < k / TILE_SIZE; ++a) {
        
        // Load the tiles into shared memory
        Ashared[threadY][threadX] = A[threadRow * k + (a * TILE_SIZE+ threadX)];
        Bshared[threadY][threadX] = B[(a * TILE_SIZE + threadX) * k + threadColumn];

        __syncthreads();

        for (int b = 0; b < TILE_SIZE; ++b) {
            calcVal += Ashared[threadY][b] * Bshared[b][threadX];

            __syncthreads();
        }

        C[threadRow * k + threadColumn] = calcVal;

    }

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    // I think that m holds the width 

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE



}


